// Very minimal skeleton for the kernel


#include <hip/hip_runtime.h>
#include <stdio.h>

extern "C" __global__ void convolute (
  const double * input,
  const double * filter,
  double * convolute_output,
  int input_width,
  int filter_width,
  int convolute_width
) {

  //dot product between filter and input

  //take output sum and put into an intermediate layer

  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;
  

  //need to determine which thread im on
  // there should be 20 x 20 threads per each output node
  // the input is 100 x 100 x 1
  // this turns into a flat 10000 size vector
  // there should be 20 x 20, 5x5 filters and 20x20 5x5 areas
  // so x, y should give an integer from 0 - 399 together
  //


  double sum = 0;
  for(int fy = 0; fy < filter_width; fy++) {
    for(int fx = 0; fx < filter_width; fx++) {
      // input dimensions are 
      // 100 x 100 x 1
      // from offset - offset + 24 where offset is the number of threads and there are offsets from 0 - 399, 
      // 

      // filter dimensions are
      // 5 x 5 x 1
      // from 0 - 24 for a square matrix
      // fx * filter_width + fy


      int offset = fx + fy * filter_width;
      int convolute_offset = fx + fy * input_width;
      //  500  * y + 5 * x
      int thread_position = y * input_width * filter_width + x * filter_width;
    
      int input_index = thread_position + convolute_offset;


      //input for thread 1 should be 
      //  0 -   4 
      //100 - 104
      //200 - 204
      //300 - 304
      //400 - 404

      //input for thread 2 should be
      //  5 -   9
      //105 - 109
      //205 - 209
      //305 - 309
      //405 - 409
      
      //input for thread 21 sohuld be 
      // 500 - 504
      // 600 - 604
      // 700 - 704
      // 800 - 804
      // 900 - 904
      
      int filter_index = z * (filter_width * filter_width) + offset;
      // printf("input_index %d input_value %f \n" , input_index , input[input_index]);
    
      sum += input[input_index] * filter[filter_index];
      // printf("x: %d y: %d z: %d, fx: %d, fy: %d input_array_position: %d filter_position: %d \n", x, y, z, fx, fy, input_index, filter_index);
    }
  }

  int convolute_index = (convolute_width * convolute_width) * z + (convolute_width * y) + x;
  convolute_output[convolute_index] = sum;
  // printf("x: %d y: %d z: %d convolute_output_position %d \n", x, y, z, convolute_index);
  

}

extern "C" __global__ void relu (
  const double * convolute_output,
  double * relu_output,
  int convolute_width
) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;


  int index = z * (convolute_width * convolute_width) + y * (convolute_width) + x;
  if(convolute_output[index] < 0) {
    relu_output[index] = 0;
  }
  else{
    relu_output[index] = convolute_output[index];
  }
      
    
  
}


extern "C" __global__ void output (
  const double * relu_output,
  const double * weights,
  double * output,
  int flatten_width
) {
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y * blockDim.y + threadIdx.y;
  int z = blockIdx.z * blockDim.z + threadIdx.z;

  double sum = 0;
  for(int i = 0; i < flatten_width; i++) {
    int weight_index = (flatten_width) * z + i; 
    sum += weights[weight_index] * relu_output[i];
  }

  output[z] = sum;

}